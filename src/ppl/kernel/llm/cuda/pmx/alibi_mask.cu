#include "hip/hip_runtime.h"
// Licensed to the Apache Software Foundation (ASF) under one
// or more contributor license agreements.  See the NOTICE file
// distributed with this work for additional information
// regarding copyright ownership.  The ASF licenses this file
// to you under the Apache License, Version 2.0 (the
// "License"); you may not use this file except in compliance
// with the License.  You may obtain a copy of the License at
//
//   http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing,
// software distributed under the License is distributed on an
// "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
// KIND, either express or implied.  See the License for the
// specific language governing permissions and limitations
// under the License.

#include "ppl/kernel/llm/cuda/pmx/alibi_mask.h"
#include "cudakernel/common/common.cuh"
#include "ppl/common/log.h"

#include <hip/hip_fp16.h>

namespace ppl { namespace kernel { namespace llm { namespace cuda { namespace pmx {

template<int32_t total_heads>
__device__ float get_slope(int32_t head) {
    int32_t closest_power_of_2 = 1 << static_cast<int32_t>(floorf(log2f(static_cast<float>(total_heads))));
    if (head <= closest_power_of_2) {
        return powf(2.0, -8.0 * head / closest_power_of_2);
    } else {
        int32_t adjusted_head = 2 * (head - closest_power_of_2) - 1;
        return powf(2.0, -4.0 * adjusted_head / closest_power_of_2);
    }
}


template<int32_t VPT, int32_t TPB, bool MASK, int32_t total_heads>
__global__ void alibi_mask_kernel(
    const int64_t *seqstarts,
    const int64_t *kvstarts,
    const half *attention_mask,
    half *alibi_mask,
    int64_t seqdim, 
    int64_t kvdim
) 
{
    const int32_t bidx = blockIdx.y;
    const int32_t tid = threadIdx.x;
    const int32_t seqsbeg = seqstarts[bidx];
    const int32_t seqsend = seqstarts[bidx + 1];
    const int32_t kvbeg = kvstarts[bidx];
    const int32_t kvend = kvstarts[bidx + 1];
    const int32_t seqlen = seqsend - seqsbeg;
    const int32_t kvlen = kvend - kvbeg;
    
    int32_t total_elements = seqlen * kvlen;

    #pragma unroll
    for (int idx = threadIdx.x; idx < total_elements; idx += TPB) {
        int32_t seqpos = idx / kvlen;
        int32_t kvpos = idx % kvlen;

        float value = kvpos - kvlen - seqpos + seqlen;
        if (value <= 0.0f) {
            float slop = get_slope<total_heads>(blockIdx.x + 1);
            value = value * slop;
        } 
        else {
            value = -INFINITY;
        }
        int64_t write_idx = blockIdx.x * kvdim * seqdim +(seqsbeg + seqpos) * kvdim + kvbeg + kvpos;
        alibi_mask[write_idx] = __float2half(value);
    }
    // int32_t total_elements = seqlen * kvdim;
    // #pragma unroll
    // for (int idx = threadIdx.x; idx < total_elements; idx += TPB) {
    //     int32_t seqpos = idx / kvdim;
    //     int32_t kvpos = idx % kvdim;
    //     float value = 0.0f;
    //     int64_t write_idx = blockIdx.x * kvdim * seqdim +(seqsbeg + seqpos) * kvdim + kvpos;
    //     if (kvpos >= kvbeg && kvpos < kvend) {
    //         int32_t mask_pos = kvpos - kvbeg;
    //         value = mask_pos - kvlen - seqpos + seqlen;
    //         if (value <= 0.0f) {
    //             float slop = get_slope<total_heads>(blockIdx.x + 1);
    //             value = value * slop;
    //         } 
    //         else {
    //             value = -INFINITY;
    //         }
    //     }
    //     alibi_mask[write_idx] = __float2half(value);
    // }
    
    if (MASK) {
        int64_t total_mask_elements = seqlen * kvdim;
        const int64_t num_pack = total_mask_elements / VPT;

        half mask_local[VPT];
        half alibi_local[VPT];
        for(int32_t pack_id = tid; pack_id < num_pack; pack_id += TPB) {
            int64_t mask_idx = seqsbeg * kvdim + pack_id * VPT;
            int64_t alibi_idx = blockIdx.x * kvdim * seqdim + seqsbeg * kvdim + pack_id * VPT;
            copy<sizeof(half) * VPT>(&attention_mask[mask_idx], mask_local);
            copy<sizeof(half) * VPT>(&alibi_mask[alibi_idx], alibi_local);
            #pragma unroll
            for(int32_t it = 0; it < VPT; it++) {
                alibi_local[it] = mask_local[it] + alibi_local[it];
            }
            copy<sizeof(half) * VPT>(alibi_local, &alibi_mask[alibi_idx]);
        }
    }
}


ppl::common::RetCode alibi_mask(
    hipStream_t stream,
    const ppl::common::TensorShape* seqstarts_shape,
    const void* seqstarts,
    const ppl::common::TensorShape* kvstarts_shape,
    const void* kvstarts,
    const ppl::common::TensorShape* attention_mask_shape,
    const void* attention_mask,
    const ppl::common::TensorShape* alibi_output_shape,
    const int64_t num_heads,
    void* alibi_output
)
{
    // seqstarts (batch + 1)
    // kvstarts (batch + 1)
    // attention_mask (seqlen, kvlen)

    if (alibi_output_shape->GetDimCount() != 3) {
        LOG(ERROR) << "alibi_output's dim should be 3, however get " << alibi_output_shape->GetDimCount() << " dim 0: " << alibi_output_shape->GetDim(0);
        return ppl::common::RC_INVALID_VALUE;
    }

    const int32_t TPB = 256;
    constexpr int32_t VPT = 16 / sizeof(half);
    const int64_t batch = seqstarts_shape->GetDim(0) - 1;
    const int64_t kv_last_dim = alibi_output_shape->GetDim(2);
    const int64_t seq_last_dim = alibi_output_shape->GetDim(1);
    LOG(INFO) << "kv_last_dim " << kv_last_dim << " seq_last_dim " << seq_last_dim << " batch " << batch << " num heads " << num_heads;
    dim3 gridDim(num_heads, batch);

    if (attention_mask != nullptr) {
        switch(num_heads)
        {
            case 40:
                alibi_mask_kernel<VPT, TPB, true, 40>
                <<<gridDim, TPB, 0, stream>>>(
                    (const int64_t*)seqstarts, 
                    (const int64_t*)kvstarts, 
                    (const half*)attention_mask, 
                    (half*)alibi_output, 
                    seq_last_dim, 
                    kv_last_dim
                );
                break;
            default:
                LOG(ERROR) << "alibi mask do not support heads " << num_heads;
                return ppl::common::RC_UNSUPPORTED;
        }
    } else {
        switch(num_heads)
        {
            case 40:
                alibi_mask_kernel<VPT, TPB, false, 40>
                <<<gridDim, TPB, 0, stream>>>(
                    (const int64_t*)seqstarts, 
                    (const int64_t*)kvstarts, 
                    nullptr, 
                    (half*)alibi_output, 
                    seq_last_dim, 
                    kv_last_dim
                );
                break;
            default:
                LOG(ERROR) << "alibi mask do not support heads " << num_heads;
                return ppl::common::RC_UNSUPPORTED;
        }
    }
    return ppl::common::RC_SUCCESS;

}


}}}}}